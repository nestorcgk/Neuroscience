#include <stdio.h>
#include <hip/hip_runtime.h>
#include "magma.h"
#include "magma_lapack.h"
int main( int argc, char** argv ){
magma_init (); magma_timestr_t float gpu_time ,
//
initialize Magma
start , end;
magma_int_t magma_int_t magma_int_t m = 8192; magma_int_t mm=m*m; float *a;
float *d_a;
float *d_r;
float *d_c; magma_int_t ione = 1; magma_int_t ISEED [4] = magma_err_t err;
// changed // a-
*dwork ldwork;
// dwork - workspace // size of dwork of indices of inter- rows; a - mxm matrix // size of a, r, c mxm matrix on the host
;
*piv, info; // piv - array
{0 ,0 ,0 ,1}; alpha = 1.0;
beta = 0.0;
// //
start = get_current_time ();
magma sgetrf gpu( m, m, d a, m, piv, &info);
// d_a- // d_r- // d_c-
mxm matrix a on mxm matrix r on mxm matrix c on
the device the device the device
// seed
const float
const float
ldwork = m * magma_get_sgetri_nb(
// allocate matrices
err = magma_smalloc_cpu( &a , mm );
err = magma_smalloc( &d_a, mm );
err = magma_smalloc( &d_r, mm );
err = magma_smalloc( &d_c, mm );
err = magma_smalloc( &dwork, ldwork);// dev. mem. for ldwork piv=(magma_int_t*)malloc(m*sizeof(magma_int_t));// host mem.
// generate random matrix a // for piv lapackf77_slarnv(&ione,ISEED,&mm,a); // random a magma_ssetmatrix( m, m, a, m, d_a, m ); // copy a -> d_a magmablas_slacpy(’A’,m,m,d_a,m,d_r,m); // copy d_a -> d_r
// find the inverse matrix: a_d*X=I using the LU factorization // with partial pivoting and row interchanges computed by
// magma_sgetrf_gpu; row i is interchanged with row piv(i);
// d_a -mxm matrix; d_a is overwritten by the inverse
magma sgetri gpu(m,d a,m,piv,dwork,ldwork,&info);
gpu_time=GetTimerValue(start,end)/1e3; // Magma time magma_sgemm(’N’,’N’,m,m,m,alpha,d_a,m,d_r,m,beta,d_c,m); printf("magma_sgetrf_gpu + magma_sgetri_gpu time: %7.5f sec.\
alpha =1 beta =0 m); // workspace size
// host memory for a // device memory for a // device memory for r // device memory for c
       end = get_current_time ();
 magma_sgetmatrix( m, m, d_c, m, a, m ); printf("upper left corner of a^-1*a:\n"); magma_sprint( 4, 4, a, m );
free(a);
free(piv);
\n",gpu_time);
// part of a^-1*a // free host memory // free host memory
 
4.3 LU decomposition and solving general linear systems 144
 
magma_free(d_a); magma_free(d_r); magma_free(d_c); magma_finalize (); return 0;
}