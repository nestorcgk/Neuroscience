#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <string>
#include <sstream>
#include <vector>
#include <utility>
#include <boost/tokenizer.hpp>
#include "kCSDAk.h"


using namespace std;
using namespace boost;
	//out: datos salida; in: datos entrada; block: tamaño del bloque; electrodes:numElectrodos;matdim dimension de la b0; origin:coord origen=y=x
	//Kernel parameters: K output matrix, Bj integral matrix (Potoworowski)
__global__ void calculateK(float * d_out, float * d_in,float * d_jlist, float * d_klist,int block, int electrodes, int matdim, int origin){
	int k = block * blockIdx.x + threadIdx.x; //column
    int j = block * blockIdx.y + threadIdx.y; //row

    if(j >= k && j < electrodes) 
    {
	    float sum = 0;
	    //Ver lo de los indices matriz julia vs c++
	    for (int l = 0; l < electrodes; ++l)
	    {
	    	//Coordenadas funcionan igual con desfase [1]
	    	//CoordenadasTotal[j]
	    	int xj1 = (int) d_jlist[j];//(int) ceil((double) j/ (double) matdim);
	    	int xj2 = (int) d_klist[j];//j % matdim;
	    	//CoordenasTodal[k]
	    	int xk1 = (int) d_jlist[k];//(int) ceil((double) k/ (double) matdim);
	    	int xk2 = (int) d_klist[k];//k % matdim;
	    	//Coordenastotal[l]
	    	int xl1 = (int) d_jlist[l];//(int) ceil((double) l/ (double) matdim);
	    	int xl2 = (int) d_klist[l];//l % matdim;
	    	//Matrix bj is stored as an array: Col + Row*dim;
	    	//xk-xl+const
	    	int idx1 = xk2-xl2+origin + (xk1-xl1+origin)*matdim;
	    	//xj-xl+const	
	    	int idx2 = xj2-xl2+origin + (xj1-xl1+origin)*matdim;
	    	sum += d_in[idx1] * d_in[idx2];//d_in[xj2 + xj1*matdim];  
	    }
	    //Equivalente a d_out[j,k] = sum
    	d_out[k + electrodes*j] = sum;
    	d_out[j + electrodes*k] = sum;
    }
}

void readData(string name, float* data){
	ifstream myfile (name);
  	//std::vector<float> data(127*127);
	//cout.precision(17);
	int i = 0;
	std::string line;

	if(myfile.is_open())
	{
		char_separator<char> sep("\t");
	while(getline(myfile, line)) 
	{
		tokenizer<char_separator<char>> tokens(line, sep);
	    for (const auto& t : tokens) 
	    {
	    		data[i] = stod(t);
				i++;
		}	    
	}
	myfile.close();
	}
	}

void writeData(float* data, int matdim){
	remove( "K.dat" );
	std::ofstream output("K.dat");
	for (int j = 0; j < matdim; ++j)
	{
		for (int k = 0; k < matdim; ++k)
		{
			output << data[k + j*matdim] << "\t";
		}
		output << endl;
	}
}	

void genCoords(float* jlist, float* klist,int matdim){
	int i = 0;
	for (int j = 0; j < matdim; ++j)
	{
		for (int k = 0; k < matdim; ++k)
		{
			jlist[i] = j;
			klist[i] = k;
			i++;
		}
	}
}

void readElec(string name, float* jlist, float* klist, int matdim){
	ifstream myfile (name);
  	//std::vector<float> data(127*127);
	//cout.precision(17);
	int i = 0;
	std::string line;

	if(myfile.is_open())
	{
		char_separator<char> sep("\t");
	while(getline(myfile, line)) 
	{
		tokenizer<char_separator<char>> tokens(line, sep);
		bool isj = true;
	    for (const auto& t : tokens) 
	    {		
	    		if(isj)
	    		{
	    			jlist[i] = stod(t);
	    			isj = false;
	    		}else{
	    			klist[i] = stod(t);
	    			i++;
	    			isj = true;
	    		}
		}	    
	}
	myfile.close();
	}
}


int main(int argc, char ** argv) {
	const int BLOCK_SIZE = 8;
	std::istringstream iss( argv[3] );
    int val;
    iss >> val;
	const int ELECTRODES = val;
	const int ORIGIN = 63;

	// const float* in, float* out
	const int DATA_SIZE_IN = 127*127;
	const int DATA_SIZE_OUT = 4096*4096;
	const int ARRAY_BYTES_DR = DATA_SIZE_IN * sizeof(float);
	const int ARRAY_BYTES_H = DATA_SIZE_OUT * sizeof(float);
	const int J_LIST_SIZE_IN = 4096;
	const int K_LIST_SIZE_IN = 4096;
	const int ARRAY_BYTES_J_LIST = J_LIST_SIZE_IN * sizeof(float);
	const int ARRAY_BYTES_K_LIST = K_LIST_SIZE_IN * sizeof(float);

	std::vector<float> data(DATA_SIZE_IN);
	std::vector<float> result(DATA_SIZE_OUT);

	//Generate list of proper electrodes
	std::vector<float> jlist(4096);
	std::vector<float> klist(4096);
	genCoords(jlist.data(), klist.data(),64);

	// declare GPU memory pointers
	float * d_in;
	float * d_out;
	float * d_jlist;
	float * d_klist;

	//Read Data
	readData(argv[1],data.data());
	readElec(argv[2],jlist.data(),klist.data(),ELECTRODES);
	//Sí lee correctamente
	/*
	for (int i = 0; i < jlist.size(); ++i)
	{
		cout << jlist[i] << "\t"<< klist[i] << "\n";
	}
	*/
	
	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES_DR);
	hipMalloc((void**) &d_out, ARRAY_BYTES_H);
	hipMalloc((void**) &d_jlist, ARRAY_BYTES_J_LIST);
	hipMalloc((void**) &d_klist, ARRAY_BYTES_K_LIST);




	// transfer the array to the GPU
	hipMemcpy(d_in, data.data(), ARRAY_BYTES_DR, hipMemcpyHostToDevice);
	hipMemcpy(d_jlist, jlist.data(), ARRAY_BYTES_J_LIST, hipMemcpyHostToDevice);
	hipMemcpy(d_klist, klist.data(), ARRAY_BYTES_K_LIST, hipMemcpyHostToDevice);

	//Set gridSize and BlockSize
	const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);  
    const dim3 gridSize(ceil(4096/ (double) BLOCK_SIZE), ceil(4096/(double) BLOCK_SIZE), 1);

	// launch the kernel
	calculateK<<<gridSize, blockSize>>>(d_out, d_in, d_jlist , d_klist, BLOCK_SIZE, ELECTRODES, 127, ORIGIN);
	
	// copy back the result array to the CPU
	hipMemcpy(result.data(), d_out, ARRAY_BYTES_H, hipMemcpyDeviceToHost);

	// writeData in file
	writeData(result.data(), ELECTRODES);
	
	hipFree(d_in);
	hipFree(d_out);

	return 0;
	
}