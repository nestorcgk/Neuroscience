#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <utility>
#include <boost/tokenizer.hpp>
#include "kCSDAk.h"


using namespace std;
using namespace boost;

	//Kernel parameters: K output matrix, Bj integral matrix (Potoworowski)
__global__ void calculateK(float * d_out, float * d_in,int block, int electrodes, int matdim ){
	int j = block * blockIdx.x + threadIdx.x; //row
    int k = block * blockIdx.y + threadIdx.y; //column

    /*
    float sum = 0;
    //Ver lo de los indices matriz julia vs c++
    for (int l = 0; l < ELECTRODES; ++l)
    {
    	int xj1 = (int) ceil((double) j/ (double) MATRIX_DIM);
    	int xj2 = j % MATRIX_DIM;
    	int xk1 = (int) ceil((double) k/ (double) MATRIX_DIM);
    	int xk2 = k % MATRIX_DIM;
    	int xl1 = (int) ceil((double) l/ (double) MATRIX_DIM);
    	int xl2 = l % MATRIX_DIM;
    	//Matrix bj is stored as an array: x + nx*y;
    	int idx1 = xk1-xl1+ORIGIN + MATRIX_DIM*(xk2-xl2+ORIGIN);	
    	int idx2 = xj1-xl1+ORIGIN + MATRIX_DIM*(xj2-xl2+ORIGIN);
    	sum +=  d_in[idx1]* d_in[idx2];
    }
    //d_out[j + MATRIX_DIM*k] = sum;
    */
    if(j < electrodes && k < electrodes) //j <= k &&
    {
    	d_out[j + electrodes*k] = j;
    }
    
}

void readData(string name, float* data){
	ifstream myfile (name);
  	//std::vector<float> data(127*127);
	//cout.precision(17);
	int i = 0;
	std::string line;

	if(myfile.is_open())
	{
		char_separator<char> sep("\t");
	while(getline(myfile, line)) 
	{
		tokenizer<char_separator<char>> tokens(line, sep);
	    for (const auto& t : tokens) 
	    {
	    		data[i] = stod(t);
				i++;
		}	    
	}
	myfile.close();
	}
	}

int main(int argc, char ** argv) {
	const int  BLOCK_SIZE = 8;
	const int ELECTRODES = 128;
	const int MATRIX_DIM = 64;
	//const int ORIGIN = 64;

	// const float* in, float* out
	const int DATA_SIZE_IN = 127*127;
	const int DATA_SIZE_OUT = 4096*4096;
	const int ARRAY_BYTES_DR = DATA_SIZE_IN * sizeof(float);
	const int ARRAY_BYTES_H = DATA_SIZE_OUT * sizeof(float);
	std::vector<float> data(DATA_SIZE_IN);
	std::vector<float> result(DATA_SIZE_OUT);

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	//Read Data
	readData(argv[1],data.data());
	/* Sí lee correctamente
	for (int i = 0; i < data.size(); ++i)
	{
		cout << data[i] << "\n";
	}
	*/

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES_DR);
	hipMalloc((void**) &d_out, ARRAY_BYTES_H);

	// transfer the array to the GPU
	hipMemcpy(d_in, data.data(), ARRAY_BYTES_DR, hipMemcpyHostToDevice);
	//Set gridSize and BlockSize
	const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);  
    const dim3 gridSize(ceil(ELECTRODES/ (double) BLOCK_SIZE), ceil(ELECTRODES/(double) BLOCK_SIZE), 1);

	// launch the kernel
	calculateK<<<gridSize, blockSize>>>(d_out, d_in, BLOCK_SIZE, ELECTRODES, MATRIX_DIM);

	// copy back the result array to the CPU
	hipMemcpy(result.data(), d_out, ARRAY_BYTES_H, hipMemcpyDeviceToHost);

	// print out the resulting array
	
	for (int i =0; i < 128*128; i++) {
		cout << i << ": " << result[i] << "\n";
		
	}
	

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}