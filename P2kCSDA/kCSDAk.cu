#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define BLOCK_SIZE 8
#define ELECTRODES 4096
#define MATRIX_DIM 64
#define ORIGIN 64
#define BDIM 127 

	//Kernel parameters: K output matrix, Bj integral matrix (Potoworowski)
__global__ void calculateK(float * d_out, float * d_in){
	int j = BLOCK_SIZE * blockIdx.x + threadIdx.x; //row
    int k = BLOCK_SIZE * blockIdx.y + threadIdx.y; //column
    //x=0:4095
    if (j >= ELECTRODES || k > j)
    	return;
    float sum = 0;
    //Ver lo de los indices matriz julia vs c++
    for (int l = 0; l < ELECTRODES; ++l)
    {
    	int xj1 = (int) ceil((double) j/ (double) MATRIX_DIM);
    	int xj2 = j % MATRIX_DIM;
    	int xk1 = (int) ceil((double) k/ (double) MATRIX_DIM);
    	int xk2 = k % MATRIX_DIM;
    	int xl1 = (int) ceil((double) l/ (double) MATRIX_DIM);
    	int xl2 = l % MATRIX_DIM;
    	//Matrix bj is stored as an array: x + nx*y;
    	int idx1 = xk1-xl1+ORIGIN + MATRIX_DIM*(xk2-xl2+ORIGIN);	
    	int idx2 = xj1-xl1+ORIGIN + MATRIX_DIM*(xj2-xl2+ORIGIN);
    	sum +=  d_in[idx1]* d_in[idx2];
    }

    d_out[j + MATRIX_DIM*k] = sum;
}

int main(int argc, char ** argv) {
	// const float* in, float* out
	const int DATA_SIZE_IN = 127*127;
	const int DATA_SIZE_OUT = 4096*4096;
	const int ARRAY_BYTES_DR = DATA_SIZE_IN * sizeof(float);
	const int ARRAY_BYTES_H = DATA_SIZE_OUT * sizeof(float);

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES_DR);
	hipMalloc((void**) &d_out, ARRAY_BYTES_H);

	// transfer the array to the GPU
	hipMemcpy(d_in, in, ARRAY_BYTES_DR, hipMemcpyHostToDevice);
	//Set gridSize and BlockSize
	const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);  
    const dim3 gridSize(ceil(ELECTRODES/ (double) BLOCK_SIZE), ceil(ELECTRODES/(double) BLOCK_SIZE), 1);

	// launch the kernel
	calculateK<<<gridSize, blockSize>>>(d_out, d_in);

	// copy back the result array to the CPU
	hipMemcpy(out, d_out, ARRAY_BYTES_H, hipMemcpyDeviceToHost);

	// print out the resulting array
	for (int i =0; i < DATA_SIZE_OUT; i++) {
		printf("%f", out[i]);
		printf("\n");
	}

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}