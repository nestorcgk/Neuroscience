#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <fstream>
#include <iostream>
#include <string>
#include <vector>
#include <utility>
#include <boost/tokenizer.hpp>
#include "kCSDAk.h"


using namespace std;
using namespace boost;
	//out: datos salida; in: datos entrada; block: tamaño del bloque; electrodes:numElectrodos;matdim dimension de la b0; origin:coord origen=y=x
	//Kernel parameters: K output matrix, Bj integral matrix (Potoworowski)
__global__ void calculateK(float * d_out, float * d_in,int block, int electrodes, int matdim, int origin){
	int j = block * blockIdx.x + threadIdx.x; //row
    int k = block * blockIdx.y + threadIdx.y; //column

    if(k <= j && j < electrodes) 
    {
	    float sum = 0;
	    //Ver lo de los indices matriz julia vs c++
	    for (int l = 0; l < electrodes; ++l)
	    {
	    	int xj1 = (int) ceil((double) j/ (double) matdim);
	    	int xj2 = j % matdim;
	    	int xk1 = (int) ceil((double) k/ (double) matdim);
	    	int xk2 = k % matdim;
	    	int xl1 = (int) ceil((double) l/ (double) matdim);
	    	int xl2 = l % matdim;
	    	//Matrix bj is stored as an array: x + nx*y;
	    	int idx1 = xk1-xl1+origin + matdim*(xk2-xl2+origin);	
	    	int idx2 = xj1-xl1+origin + matdim*(xj2-xl2+origin);
	    	sum +=  d_in[idx1]* d_in[idx2];
	    }
	  
    	d_out[j + electrodes*k] = sum;
    }
    
}

void readData(string name, float* data){
	ifstream myfile (name);
  	//std::vector<float> data(127*127);
	//cout.precision(17);
	int i = 0;
	std::string line;

	if(myfile.is_open())
	{
		char_separator<char> sep("\t");
	while(getline(myfile, line)) 
	{
		tokenizer<char_separator<char>> tokens(line, sep);
	    for (const auto& t : tokens) 
	    {
	    		data[i] = stod(t);
				i++;
		}	    
	}
	myfile.close();
	}
	}

void writeData(float* data, int matdim){
	std::ofstream output("K.dat");
	for (int j = 0; j < matdim; ++j)
	{
		for (int k = 0; k < matdim; ++k)
		{
			output << data[k + j*matdim] << "\t";
		}
		output << endl;
	}
}	

int main(int argc, char ** argv) {
	const int  BLOCK_SIZE = 8;
	const int ELECTRODES = 128;
	const int MATRIX_DIM = 64;
	const int ORIGIN = 64;

	// const float* in, float* out
	const int DATA_SIZE_IN = 127*127;
	const int DATA_SIZE_OUT = 4096*4096;
	const int ARRAY_BYTES_DR = DATA_SIZE_IN * sizeof(float);
	const int ARRAY_BYTES_H = DATA_SIZE_OUT * sizeof(float);
	std::vector<float> data(DATA_SIZE_IN);
	std::vector<float> result(DATA_SIZE_OUT);

	// declare GPU memory pointers
	float * d_in;
	float * d_out;

	//Read Data
	readData(argv[1],data.data());
	/* Sí lee correctamente
	for (int i = 0; i < data.size(); ++i)
	{
		cout << data[i] << "\n";
	}
	*/

	// allocate GPU memory
	hipMalloc((void**) &d_in, ARRAY_BYTES_DR);
	hipMalloc((void**) &d_out, ARRAY_BYTES_H);

	// transfer the array to the GPU
	hipMemcpy(d_in, data.data(), ARRAY_BYTES_DR, hipMemcpyHostToDevice);
	//Set gridSize and BlockSize
	const dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE, 1);  
    const dim3 gridSize(ceil(ELECTRODES/ (double) BLOCK_SIZE), ceil(ELECTRODES/(double) BLOCK_SIZE), 1);

	// launch the kernel
	calculateK<<<gridSize, blockSize>>>(d_out, d_in, BLOCK_SIZE, ELECTRODES, MATRIX_DIM, ORIGIN);

	// copy back the result array to the CPU
	hipMemcpy(result.data(), d_out, ARRAY_BYTES_H, hipMemcpyDeviceToHost);

	// writeData in file
	writeData(result.data(), ELECTRODES);
	
	
	

	hipFree(d_in);
	hipFree(d_out);

	return 0;
}